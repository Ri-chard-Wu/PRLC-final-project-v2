#include "hip/hip_runtime.h"

#include <cstdlib>
#include <iostream>
#include <iomanip>
#include "kissrandom.h"
#include "annoylib.h"
#include <chrono>
#include <algorithm>
#include <map>
#include <random>




int f = 786;
int n = 1000000;
int n_trees = 5;
char *filename = "AnnoyGPU-1e6.tree";

int search_multiplier = 20;
int GPU_BUILD_MAX_ITEM_NUM = 500000;






using namespace Annoy;


int fill_item(char *filename, int f=40, int n=1000000){

	AnnoyIndex<int, float, Angular, Kiss32Random, AnnoyIndexGPUBuildPolicy> t(f);

	std::default_random_engine generator;

	t.on_disk_build(filename);
	
	for(int i = 0; i < n; ++i){ // n: number of vectors

		float *vec = (float *) malloc( f * sizeof(float) );

		float mean = 0.0;
		float std = 1.0;		
		std::normal_distribution<float> distribution(mean, std);
		
		for(int z = 0; z < f; ++z){ // f: vector dim.
			vec[z] = (distribution(generator));
		}

		t.add_item(i, vec);

		if(i % 1024 == 0){
			std::cout << "Loading objects ...\t object: "
					<< i+1 
					<< "\tProgress:"
					<< std::fixed 
					<< std::setprecision(2) 
					<< (float) i / (float)(n + 1) * 100 
					<< "%\r";		
		}	  
	}
}


void load_item(AnnoyIndex<int, float, Angular, Kiss32Random, AnnoyIndexGPUBuildPolicy>& t, char *filename, int n){
	
	// t.load(filename);
	t.load_items(filename, n);

}


void build_index(AnnoyIndex<int, float, Angular, Kiss32Random, AnnoyIndexGPUBuildPolicy>& t, int n_trees){


	std::chrono::high_resolution_clock::time_point t_start, t_end;


	std::cout << std::endl;
	// std::cout << "Building index num_trees = 2 * num_features ...\n\n\n";

	t_start = std::chrono::high_resolution_clock::now();
	t.build(n_trees);
	t_end = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::\
				duration_cast<std::chrono::seconds>( t_end - t_start ).count();
	std::cout << " Done in "<< duration << " secs." << std::endl;

	// std::cout << "Saving index ...";
	// t.save("precision.tree");
	// std::cout << " Done" << std::endl;
}



int precision_test(AnnoyIndex<int, float, Angular, Kiss32Random, AnnoyIndexGPUBuildPolicy>& t, 
			int f=40, int n=1000000, int n_trees=80){

	std::chrono::high_resolution_clock::time_point t_start, t_end;

	std::vector<int> limits = {10, 100, 1000, 10000};
	int K=10;
	int prec_n = 10;

	std::map<int, double> prec_sum;
	std::map<int, double> time_sum;
	std::vector<int> closest;

	for(std::vector<int>::iterator it = limits.begin(); 
									it != limits.end(); ++it){
		prec_sum[(*it)] = 0.0;
		time_sum[(*it)] = 0.0;
	}


	for(int i = 0; i < prec_n; ++i){

		int j = rand() % n;

		t.get_nns_by_item(j, K, n, &closest, nullptr);

		std::vector<int> toplist;
		std::vector<int> intersection;

		for(std::vector<int>::iterator limit = limits.begin(); 
										limit!=limits.end(); ++limit){

			t_start = std::chrono::high_resolution_clock::now();

			//search_k defaults to "n_trees * (*limit)" (which is  << n) if not provided (pass -1).
			t.get_nns_by_item(j, (*limit), search_multiplier * n_trees * (*limit), &toplist, nullptr); 

			
			t_end = std::chrono::high_resolution_clock::now();
			auto duration = std::chrono::duration_cast\
					<std::chrono::milliseconds>( t_end - t_start ).count();

			std::sort(closest.begin(), closest.end(), std::less<int>());
			std::sort(toplist.begin(), toplist.end(), std::less<int>());

			intersection.resize(std::max(closest.size(), toplist.size()));
			
			std::vector<int>::iterator it_set = \
				std::set_intersection(closest.begin(), closest.end(), \
					toplist.begin(), toplist.end(), intersection.begin());

			intersection.resize(it_set-intersection.begin());

			int found = intersection.size();
			double hitrate = found / (double) K;
			prec_sum[(*limit)] += hitrate;
			time_sum[(*limit)] += duration;
 
			vector<int>().swap(intersection);
			vector<int>().swap(toplist);
		}


		closest.clear(); 
		vector<int>().swap(closest);
	}

	for(std::vector<int>::iterator limit = limits.begin(); limit!=limits.end(); ++limit){
		std::cout << "limit: " << (*limit) << "\tprecision: "<< std::fixed 
			<< std::setprecision(2) << (100.0 * prec_sum[(*limit)] / prec_n)
					<< "% \tavg. time: "<< std::fixed<< std::setprecision(6) 
					<< (time_sum[(*limit)] / prec_n) * 1e-04 << "s" << std::endl;
	}


	std::cout << "\nDone" << std::endl;
	return 0;
}



int precision(int f=40, int n=1000000, int n_trees=80){

	std::chrono::high_resolution_clock::time_point t_start, t_end;

	std::default_random_engine generator;

	AnnoyIndex<int, float, Angular, Kiss32Random, AnnoyIndexGPUBuildPolicy> t(f);


	char *filename = "test_disk_build.tree";
	t.on_disk_build(filename);

	// std::cout << "Building index ..." << std::endl;


	
	for(int i = 0; i < n; ++i){ // n: number of vectors

		float *vec = (float *) malloc( f * sizeof(float) );

		// double mean = (double)(rand() % 10);
		// double std = (double)(rand() % 5);
		float mean = 0.0;
		float std = 1.0;		
		std::normal_distribution<float> distribution(mean, std);
		
		for(int z = 0; z < f; ++z){ // f: vector dim.

			vec[z] = (distribution(generator));
		}

		t.add_item(i, vec);

		// std::cout << "Loading objects ...\t object: "
		// 		  << i+1 
		// 		  << "\tProgress:"
		// 		  << std::fixed 
		// 		  << std::setprecision(2) 
		// 		  << (float) i / (float)(n + 1) * 100 
		// 		  << "%\r";
		
						  
	}


	std::cout << std::endl;
	// std::cout << "Building index num_trees = 2 * num_features ...\n\n\n";

	t_start = std::chrono::high_resolution_clock::now();
	t.build(n_trees);
	t_end = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::seconds>( t_end - t_start ).count();
	std::cout << " Done in "<< duration << " secs." << std::endl;

	// std::cout << "Saving index ...";
	// t.save("precision.tree");
	// std::cout << " Done" << std::endl;


	//******************************************************


	std::vector<int> limits = {10, 100, 1000, 10000};
	int K=10;
	int prec_n = 10;

	std::map<int, double> prec_sum;
	std::map<int, double> time_sum;
	std::vector<int> closest;

	//init precision and timers map
	for(std::vector<int>::iterator it = limits.begin(); 
									it != limits.end(); ++it){
		prec_sum[(*it)] = 0.0;
		time_sum[(*it)] = 0.0;
	}


	// test precision with `prec_n` random number.
	for(int i = 0; i < prec_n; ++i){

		// select a random node
		int j = rand() % n;

		// std::cout << "finding nbs for " << j << std::endl;

		// getting the K closest
		// search all n nodes, very slow but most accurate achievable.
		t.get_nns_by_item(j, K, n, &closest, nullptr);

		std::vector<int> toplist;
		std::vector<int> intersection;

		for(std::vector<int>::iterator limit = limits.begin(); 
										limit!=limits.end(); ++limit){

			t_start = std::chrono::high_resolution_clock::now();

			//search_k defaults to "n_trees * (*limit)" 
				// (which is  << n) if not provided (pass -1).
			t.get_nns_by_item(j, (*limit), (size_t) -1, &toplist, nullptr); 

			
			t_end = std::chrono::high_resolution_clock::now();
			auto duration = std::chrono::duration_cast\
					<std::chrono::milliseconds>( t_end - t_start ).count();

			std::sort(closest.begin(), closest.end(), std::less<int>());
			std::sort(toplist.begin(), toplist.end(), std::less<int>());

			intersection.resize(std::max(closest.size(), toplist.size()));
			
			std::vector<int>::iterator it_set = \
				std::set_intersection(closest.begin(), closest.end(), \
					toplist.begin(), toplist.end(), intersection.begin());

			intersection.resize(it_set-intersection.begin());

			int found = intersection.size();
			double hitrate = found / (double) K;
			prec_sum[(*limit)] += hitrate;
			time_sum[(*limit)] += duration;
 
			vector<int>().swap(intersection);
			vector<int>().swap(toplist);
		}


		closest.clear(); 
		vector<int>().swap(closest);
	}

	for(std::vector<int>::iterator limit = limits.begin(); limit!=limits.end(); ++limit){
		std::cout << "limit: " << (*limit) << "\tprecision: "<< std::fixed 
			<< std::setprecision(2) << (100.0 * prec_sum[(*limit)] / prec_n)
					<< "% \tavg. time: "<< std::fixed<< std::setprecision(6) 
					<< (time_sum[(*limit)] / prec_n) * 1e-04 << "s" << std::endl;
	}


	std::cout << "\nDone" << std::endl;
	return 0;
}



// #############################################################
/*
gpu, 1e6, 5, no partition  -------------------------------------------

 Done in 20 secs.
limit: 10       precision: 11.00%       avg. time: 0.000100s
limit: 100      precision: 11.00%       avg. time: 0.000210s
limit: 1000     precision: 12.00%       avg. time: 0.001930s
limit: 10000    precision: 24.00%       avg. time: 0.019520s


gpu, 1e6, 5, partition into 4 -------------------------------------------

 Done in 19 secs.
limit: 10       precision: 4.00%        avg. time: 0.000130s
limit: 100      precision: 5.00%        avg. time: 0.000300s
limit: 1000     precision: 11.00%       avg. time: 0.001580s
limit: 10000    precision: 21.00%       avg. time: 0.016340s

gpu, 1e6, 5, partition into 4, search 5x -------------------------------------------

 Done in 19 secs.
limit: 10       precision: 3.00%        avg. time: 0.000140s
limit: 100      precision: 6.00%        avg. time: 0.000640s
limit: 1000     precision: 15.00%       avg. time: 0.006720s
limit: 10000    precision: 45.00%       avg. time: 0.064340s

gpu, 1e6, 5, partition into 4, search 20x -------------------------------------------

 Done in 19 secs.
limit: 10       precision: 9.00%        avg. time: 0.000330s
limit: 100      precision: 16.00%       avg. time: 0.002510s
limit: 1000     precision: 28.00%       avg. time: 0.026500s
limit: 10000    precision: 100.00%      avg. time: 0.232380s

gpu, 1e6, 5, no partition, search 20x -------------------------------------------

 Done in 20 secs.
limit: 10       precision: 11.00%       avg. time: 0.000360s
limit: 100      precision: 15.00%       avg. time: 0.003350s
limit: 1000     precision: 32.00%       avg. time: 0.031870s
limit: 10000    precision: 100.00%      avg. time: 0.255850s
*/

// #############################################################

/*

cpu, 5e6, 5 -------------------------------------------

 Done in 1070 secs.
limit: 10       precision: 10.00%       avg. time: 0.000160s
limit: 100      precision: 10.00%       avg. time: 0.000210s
limit: 1000     precision: 11.00%       avg. time: 0.002200s
limit: 10000    precision: 17.00%       avg. time: 0.022570s


gpu, 5e6, 5, no partition -------------------------------------

 Done in 125 secs.
limit: 10       precision: 1.00%        avg. time: 0.000180s
limit: 100      precision: 7.00%        avg. time: 0.000700s
limit: 1000     precision: 12.00%       avg. time: 0.006390s
limit: 10000    precision: 23.00%       avg. time: 0.066370s


gpu, 5e6, 5, partition into 5, search 20x -------------------------------------------

 Done in 112 secs.
limit: 10       precision: 2.00%        avg. time: 0.000410s
limit: 100      precision: 11.00%       avg. time: 0.003170s
limit: 1000     precision: 16.00%       avg. time: 0.031250s
limit: 10000    precision: 39.00%       avg. time: 0.309640s

*/







// -------------------------------------------
// gpu, create_split() no parallel, 1e6 nodes, 5 trees: Done in 25 secs.
// gpu, create_split() with parallel, 1e6 nodes, 5 trees: Done in 22 secs.

int main(int argc, char **argv) {


	// fill_item("AnnoyGPU-5e6.tree", f, n);
	
	AnnoyIndex<int, float, Angular, Kiss32Random, AnnoyIndexGPUBuildPolicy> t(f);
	load_item(t, filename, n);

	t.GPU_BUILD_MAX_ITEM_NUM = GPU_BUILD_MAX_ITEM_NUM;

	build_index(t, n_trees);
	precision_test(t, f, n, n_trees);




	return EXIT_SUCCESS;
}